#include "hip/hip_runtime.h"
#include "nbodysim.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void leapfrog_integrate(simdata_t *d_sdata, float *d_acceleration,
                                   float dt, bool before_accel_update) {
  int idx = threadIdx.x + blockIdx.x * 1024;
  if (idx >= d_sdata->nparticles) return;
  float *d_pos = simdata_pos_ptr(d_sdata, idx);
  float *d_vel = simdata_vel_ptr(d_sdata, idx);
  float *d_accel = d_acceleration + d_sdata->posdim * idx;

  for (int i = 0; i < d_sdata->posdim; i++) {
    d_vel[i] += d_accel[i] * dt * 0.5;
    if (before_accel_update) {
      d_pos[i] += d_vel[i] * dt;
    }
  }
}

__global__ void euler_integrate(simdata_t *d_sdata, float *d_acceleration,
                                float dt) {
  int idx = threadIdx.x + blockIdx.x * 1024;
  if (idx >= d_sdata->nparticles) return;
  float *d_pos = simdata_pos_ptr(d_sdata, idx);
  float *d_vel = simdata_vel_ptr(d_sdata, idx);
  float *d_accel = d_acceleration + d_sdata->posdim * idx;

  for (int i = 0; i < d_sdata->posdim; i++) {
    d_pos[i] += d_vel[i] * dt + 0.5 * d_accel[i] * dt * dt;
    d_vel[i] += d_accel[i] * dt;
  }
}
